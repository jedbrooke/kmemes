#include "hip/hip_runtime.h"
#include "kmeans.h"
#include <cstdio>
#include <>




uint sum(uint* a, uint n){
    uint sum = 0;
    for (uint i = 0; i < n; i++){
        sum += a[i];
    }
    return sum;
}


#define CUDA_CALL(x) do {if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);}} while(0)

// Zsums is k * w * h * d, the sum of all observations in a group
// Z is Zsum/l, the average of all observations in a group
// counts is the number of observations in a group
// k is the number of groups

__global__
void divide_Zsums(const uint* Zsums, feature_type* Z, const uint* counts) {
    const size_t w = blockDim.x * gridDim.x;
    const size_t h = blockDim.y * gridDim.y;
    const size_t d = blockDim.z;
    const size_t posx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t posy = blockIdx.y * blockDim.y + threadIdx.y;
    
    __shared__ size_t counts_shared[k];
    
    // the 0th thread in each block loads the shared mem.
    // TODO: see if this is even faster than just not using shared mem.
    if (threadIdx.x == 0 and threadIdx.y == 0 and threadIdx.z == 0) {
#pragma unroll
        for(size_t i = 0; i < k; i++){
            counts_shared[i] = counts[i];
        }
    }
    __syncthreads();

    for(size_t i = 0; i < k; i++) {
        size_t idx = i * (w * h * d) + (posy * w * d) + (posx * d) + threadIdx.z;
        Z[idx] = Zsums[idx] / counts_shared[i];
    }

}

// Z is k * h * w * d
// Z holds the group representatives
// observations is N * h * w * d
// group_distances is N * k, distance to each group for each observation
// h,w,d are the dimensions of each observation
// d is assumed to be 3

// each block handles observation at a time
// each block is k * rowSize
// each thread handles 1 feature and 1 group


__global__
void calculate_group_distances(const unsigned char* Z, const unsigned char* observations, uint* group_distances, const size_t h, const size_t w, const size_t N) {
    const size_t index = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + blockIdx.x;
    const size_t stride = gridDim.x * gridDim.y * gridDim.z;
    
    for(size_t i = index; i < N; i+= stride) {
        __shared__ uint distances[k][ROW_SIZE];

        distances[threadIdx.x][threadIdx.y] = 0;

        const size_t observation_offset = i * h * w * 3;
        
        for(size_t posy = 0; posy < h; posy++) {
            for(size_t posx = 0; posx < w; posx += ROW_SIZE) {
                
                __shared__ u_int32_t row[ROW_SIZE];
                if(threadIdx.x == 0) {
                    const size_t pixel_offset = observation_offset + (posy * w * 3) + ((posx + threadIdx.y) * 3);
                    row[threadIdx.y] = (0x0000) | (observations[pixel_offset + 0] << 16)
                                                | (observations[pixel_offset + 1] <<  8)
                                                | (observations[pixel_offset + 2] <<  0);
                }

                __syncthreads();
                const size_t pixel_offset = (threadIdx.x * h * w * 3) + (posy * w) + ((posx + threadIdx.y) * 3);
                u_int32_t group_rep = (0x0000)  | (Z[pixel_offset + 0] << 16)
                                                | (Z[pixel_offset + 1] <<  8)
                                                | (Z[pixel_offset + 2] <<  0);
                // Sum(|Z[i] - Observation[i]|)
                // original algo squares each difference, but this should work too, might just converge slower
                distances[threadIdx.x][threadIdx.y] += __vsadu4(group_rep, row[threadIdx.y]);
            }
        }
        __syncthreads();
        if(threadIdx.y == 0) {
            uint distance = 0;
            for(size_t group = 0; group < k; group++) {
                distance += distances[threadIdx.x][group];
            }
            group_distances[(i*k) + threadIdx.x] = distance;
        }
    }
}

// f = h * w * d, the number of features in each observation
// Z is k * f
// Z holds the group representatives
// observations is N * f
// group_distances is N * k, distance to each group for each observation
// h,w,d are the dimensions of each observation

// each block handles 1 observation
// each block handles observation at a time
// each block is k * ROW_SIZE
// each thread handles 1 feature and 1 group

__global__
void calculate_group_squared_distances(const unsigned char* Z, const unsigned char* observations, uint* group_distances, const size_t f, const size_t N) {
    const size_t index = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + blockIdx.x;
    const size_t stride = gridDim.x * gridDim.y * gridDim.z;

    for(size_t i = index; i < N; i+= stride) {
        
        __shared__ uint distances[k][ROW_SIZE];
        distances[threadIdx.x][threadIdx.y] = 0;
        
        const size_t observation_offset = i * f;
        
        for(size_t j = 0; j < f; j+= ROW_SIZE) {
            __shared__ feature_type chunk[ROW_SIZE];
            if(threadIdx.x == 0) {
                chunk[threadIdx.y] = observations[observation_offset + j + threadIdx.y];
            }
            __syncthreads();
            // int temp = Z[threadIdx.x * j] - chunk[threadIdx.y];
            distances[threadIdx.x][threadIdx.y] += abs(Z[threadIdx.x * j] - chunk[threadIdx.y]);

        }
        __syncthreads();
        if(threadIdx.y == 0) {
            uint distance = 0;
            for(size_t group = 0; group < k; group++) {
                distance += distances[threadIdx.x][group];
            }
            group_distances[(i * k) + threadIdx.x] = distance;
        } 
    }

}
// group_distances is N * k
// min_distances is N *
// groups is N *
// each thread handles 1 observation

__global__
void find_min_distance_and_group(const uint* group_distances, uint* min_distances, uint* groups, size_t N) {
    const size_t index = (blockIdx.x * blockDim.x) + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;
    for(size_t i = index; i < N; i+=stride) {
        uint min_distance = INT_MAX;
        int group = -1;
        for(size_t j = 0; j < k; j++) {
            if (group_distances[(i * k) + j] < min_distance) {
                min_distance = group_distances[(i * k) + j];
                group = j;
            }
        }   
        min_distances[i] = min_distance;
        groups[i] = group;
        
    }

}


// each thread will handle a single feature location across all observations
// each block is BLOCK_SIZExBLOCK_SIZEx3
// each x,y in a block will handle an x,y in the image
// and each thread along z will handle r,g,b respectively


// Zsum is h * w * d
// features is N * h * w * d
// groups is N, hold the group number of each observation

// goup is the target group we are summing for
// N is the number of observations

__global__
void calc_Zsums_for_group(const feature_type* observations, uint* groups, uint* Zsums, size_t group, size_t N) {
    const size_t w = blockDim.x * gridDim.x;
    const size_t h = blockDim.y * gridDim.y;
    const size_t d = blockDim.z;
    const size_t posx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const size_t posy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const size_t group_offset = group * h * w * d;
    for(size_t i = 0; i < N; i++) {
        if(groups[i] == group) {
            const size_t feature_idx = (posy * w * d) + (posx * d) + threadIdx.z;
            const size_t observation_idx = i * (h * w * d) + feature_idx;
            Zsums[group_offset + feature_idx] += observations[observation_idx];
        }
    }

}

__global__
void print_arr(uint* arr, size_t N) {
    if(threadIdx.x == 0 and blockIdx.x == 0) {
        for(size_t i = 0; i < N; i++) {
            printf("%u,",arr[i]);
        }
        printf("\n");
    }
}

observation* kmeans_gpu(observation* data, size_t N, size_t h, size_t w, size_t d) {
    
    // allocate memory for group representatives
    feature_type* Z_cu;
    CUDA_CALL(hipMalloc(&Z_cu, k * h * w * d * sizeof(feature_type)));

    // allocate memory for temporary group sums
    uint* Z_sums_cu;
    CUDA_CALL(hipMalloc(&Z_sums_cu, k * h * w * d * sizeof(uint)));

    // allocate memory to store group counts
    uint* Zcounts = (uint*) malloc(k * sizeof(uint));
    uint* Zcounts_cu;
    CUDA_CALL(hipMalloc(&Zcounts_cu, k * sizeof(uint)));


    // allocate memory on gpu to hold observation data
    feature_type* observations_cu;
    CUDA_CALL(hipMalloc(& observations_cu, N * h * w * d * sizeof(feature_type)));

    // allocate memory to store group distances
    uint* group_distances_cu;
    CUDA_CALL(hipMalloc(&group_distances_cu, N * k * sizeof(uint)));

    // allocate memory for minimum distances
    uint* min_distances_cu;
    CUDA_CALL(hipMalloc(&min_distances_cu, N * sizeof(uint)));
    uint* D = (uint*) malloc(N * sizeof(uint));
    
    // send data to gpu
    // TODO: split this up into batches of data, since we can't fit all of it in VRAM
    const size_t size = (h * w * d * sizeof(feature_type));
    for(size_t i = 0; i < N; i++) {
        CUDA_CALL(hipMemcpy(&observations_cu[(i * h * w * d)], data[i].features, size, hipMemcpyHostToDevice));
    }
    CUDA_CALL(hipDeviceSynchronize());


    float prev_j = 0;
    float Jscore = -1;
    float threshold = 1e-4;

    bool stop_looping = false;

    int max_iterations = 100;
    int iterations = 0;

    // assign initial random groups
    for(size_t i = 0; i < N; i++) {
        // non-random for now so we can get consistent results during testing
        data[i].group = i % k;
    }

    // send group assignments to GPU
    uint* groups = (uint*) malloc(N * sizeof(uint));
    uint* groups_cu;
    CUDA_CALL(hipMalloc(&groups_cu, N * sizeof(uint)));
    for(size_t i = 0; i < N; i++) {
        groups[i] = data[i].group;
    }
    CUDA_CALL(hipMemcpy(groups_cu, groups, N * sizeof(uint), hipMemcpyHostToDevice));

    // standard 1 thread per pixel component block/grid size used by many kernels
    dim3 standardBlock(BLOCK_SIZE,BLOCK_SIZE,d);
    printf("standard block: %d,%d,%d\n",standardBlock.x, standardBlock.y, standardBlock.z);
    dim3 standardGrid(h / BLOCK_SIZE, w / BLOCK_SIZE);
    printf("standard grid: %d,%d,%d\n",standardGrid.x, standardGrid.y, standardGrid.z);


    // k * ROW_SIZE blocks for group distance kernel
    dim3 rowBlock(k,ROW_SIZE);
    printf("row block grid: %d,%d,%d\n",rowBlock.x, rowBlock.y, rowBlock.z);

    // get num SMs, used for launching some kernels
    int numSMs;
    CUDA_CALL(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
    

    while(not stop_looping) {
        printf("epoch %d\n", iterations);

        CUDA_CALL(hipMemset(Z_sums_cu, 0, k * h * w * d * sizeof(uint)));
        bzero(Zcounts, k * sizeof(uint));


        // sum all the members in each group to find the averages
        // TODO: split this up into batches of data, since we can't fit all of it in VRAM

        // TODO: can we speed this up by running each k in a separate stream?
        for(size_t g = 0; g < k; g++) {
            calc_Zsums_for_group<<<standardGrid,standardBlock>>>(observations_cu, groups_cu, Z_sums_cu, g, N);
            hipDeviceSynchronize();
        }
        // printf("Zsums on gpu\n");
        // print_arr<<<1,1>>>(Z_sums_cu, k);
        
        // calculate the counts for each group
        for(size_t i = 0; i < N; i++) {
            Zcounts[groups[i]] += 1;
        }

        // printf("Zcounts:\n");
        // for(size_t i = 0; i < k; i++) {
        //     printf("%d,",Zcounts[i]);
        // }
        // printf("\n");

        // send Zcounts to the gpu
        CUDA_CALL(hipMemcpy(Zcounts_cu, Zcounts, k * sizeof(uint), hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        
        printf("Zcounts on gpu\n");
        print_arr<<<1,1>>>(Zcounts_cu, k);

        divide_Zsums<<<standardGrid,standardBlock>>>(Z_sums_cu, Z_cu, Zcounts_cu);
        hipDeviceSynchronize();

        // calculate group distances
        calculate_group_distances<<<32*numSMs,rowBlock>>>(Z_cu, observations_cu, group_distances_cu, h, w, N);
        // calculate_group_squared_distances<<<32*numSMs,rowBlock>>>(Z_cu, observations_cu, group_distances_cu, h * w * d, N);
        hipDeviceSynchronize();

        // find minimum distance for each observation and assign groups
        find_min_distance_and_group<<<32*numSMs, BLOCK_SIZE * BLOCK_SIZE>>>(group_distances_cu, min_distances_cu, groups_cu, N);
        hipDeviceSynchronize();

        // copy groups back to cpu
        CUDA_CALL(hipMemcpy(groups, groups_cu, N * sizeof(uint), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();

        // copy min distances back to host
        // TODO: since D is already in gpu mem, and it may be big enough to benefit from a parallel reduction sum
        // we could just calculate the Jscore on the gpu
        hipMemcpy(D, min_distances_cu, N * sizeof(uint), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        Jscore = abs((float) sum(D,N) / N);
        printf("J = %0.5f\n",Jscore);
        printf("Diff = %0.5f\n",fabs(Jscore - prev_j));
        printf("Thresh = %0.5f\n",(threshold * Jscore));
        iterations++;
        stop_looping = fabs(Jscore - prev_j) < fabs(threshold * Jscore) || iterations >= max_iterations;
        prev_j = Jscore;

    }

    observation* Z = (observation*) malloc(k * sizeof(observation));
    for(size_t i = 0; i < k; i++) {
        Z[i].features = (feature_type*) malloc(h * w * d * sizeof(feature_type));
        CUDA_CALL(hipMemcpyAsync(Z[i].features,&Z_cu[i * h * w * d], h * w * d * sizeof(feature_type), hipMemcpyDeviceToHost));
        Z[i].group = i;
    }
    hipDeviceSynchronize();
    return Z;
}